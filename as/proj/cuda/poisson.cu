#define FP float
#define BDIM 512
#define MAX_ITERATION 1500
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>

// handle vector operations, c = a + ceof * b
__global__ void gpu_vecop(FP *a, FP *b, FP *c, FP coef, int N) {
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  if(index < N) c[index] = a[index] + coef * b[index];
}
// handle dop product, res = dot(a, b)
__global__ void gpu_vecdot(FP *a, FP *b, FP *c, int N) {
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  if(index < N) {
    c[index] = a[index] * b[index]; 
  }
}

// handle assignment, a = b;
__global__ void gpu_vecassign(FP *a, FP *b, int N) {
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  if(index < N) a[index] = b[index]; 
}
// reset vector, a = 0
__global__ void gpu_vecreset(FP *a, int N) {
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  if(index < N) a[index] = 0.; 
}
// handle matrix times vector, c = a * b
__global__ void gpu_matmulvec(FP *a, int *row, int *col, FP *b, FP *c, int Nz) {
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  int start_idx = row[index], end_idx = row[index+1];
  for(int i=start_idx; i<end_idx; i++) {
    c[index] += a[i] * b[col[i]];
  }
}

// use CSR scheme to store the matrix
void initialize(FP *A, int *JR, int *JC, FP *b, int n, int N) {

  // initialize matrix A
  int index = 0;
  FP a[] = {-4, 20, -4};
  FP t[] = {-1, -4, -1};

  for (int grid_row = 0; grid_row < n; grid_row++) {
      int t0_grid_left = (grid_row - 1) * n, t0_grid_right = grid_row * n;
      int a_grid_left = grid_row * n, a_grid_right = (grid_row + 1) * n;
      int t1_grid_left = (grid_row + 1) * n, t1_grid_right = (grid_row + 2) * n;
      for (int row = grid_row * n, col = (grid_row - 1) * n - 1; row < (grid_row + 1) * n; row++, col++) {
          JR[row] = index;
          int base_col = col;
          //set up t0
          if (grid_row != 0) {
              for (int i = 0; i < 3; i++) {
                  int col_idx = base_col + i;
                  if (t0_grid_left <= col_idx && base_col <= col_idx && col_idx < base_col + n &&
                      col_idx < t0_grid_right) {
                      A[index] = t[i];
                      JC[index] = col_idx;
                      index++;
                  }
              }
          }
          //set up Ak
          base_col += n;
          for (int i = 0; i < 3; i++) {
              int col_idx = base_col + i;
              if (a_grid_left <= col_idx && base_col <= col_idx && col_idx < base_col + n && col_idx < a_grid_right) {
                  A[index] = a[i];
                  JC[index] = col_idx;
                  index++;
              }
          }
          base_col += n;
          //set up t1
          if (grid_row != n - 1) {
              for (int i = 0; i < 3; i++) {
                  int col_idx = base_col + i;
                  if (t1_grid_left <= col_idx && base_col <= col_idx && col_idx < base_col + n &&
                      col_idx < t1_grid_right) {
                      A[index] = t[i];
                      JC[index] = col_idx;
                      index++;
                  }
              }
          }
      }
  }

  JR[N] = index; 

  // initialize vector b
  FP num = 6.0 / (n + 1) / (n + 1);
  b[(n / 2 - 1) * n + (n / 2 - 1)] = num;
  b[(n / 2 - 1) * n + (n / 2)] = num;
  b[(n / 2) * n + (n / 2 - 1)] = num;
  b[(n / 2) * n + (n / 2)] = num;

}

void print(FP *a, int n, int m) {
	for(int i=0; i<n; i++) {
		for(int j=0; j<m; j++) {
			printf("%.6e ", a[i*m+j]);
		}
		printf("\n");
	}
}


FP reduce(FP *a, int N) {
  FP res = 0.;
  for(int i=0; i<N; i++) res += a[i];
  return res;
}

int main(int argc, char *argv[]) {

  int gpucount = 0; // Count of available GPUs
  int gpunum = 0; // Device number to use

  int n, N, iter; // matrix dimension
  int Nz_in_grid, Nz; 
  float rho, prev_rho;
  int *row, *col, *dev_row, *dev_col;
  FP *A, *b, *x, *p, *r, *q, *temp;
  FP *dev_A, *dev_b, *dev_x, *dev_p, *dev_r, *dev_q, *dev_temp;

  hipEvent_t start, stop; // using cuda events to measure time
  float elapsed_time_ms; // which is applicable for asynchronous code also
  hipError_t errorcode;

  // --------------------SET PARAMETERS AND DATA -----------------------

  errorcode = hipGetDeviceCount(&gpucount);
  if (errorcode == hipErrorNoDevice) {
    printf("No GPUs are visible\n");
    exit(-1);
  }
  else {
     printf("Device count = %d\n",gpucount);
  }

  if ((argc<2) || (argc>3)) {
    printf("Usage: matmul <matrix dim n> [<dev num>]\n");
    exit (-1);
  }

  n = atoi(argv[1]);
  N = n * n;
  Nz_in_grid = (n - 2) * 3 + 2 * 2; // number of non-zero elements in a grid
  Nz = (n - 2) * 3 * Nz_in_grid + 2 * 2 * Nz_in_grid; // number of non-zero elements
  printf("Nz: %d\n", Nz);
  row = (int *) calloc(N+1, sizeof(int));
  col = (int *) calloc(Nz, sizeof(int));
  A = (FP *) calloc(Nz, sizeof(FP));
  b = (FP *) calloc(N, sizeof(FP));
  x = (FP *) calloc(N, sizeof(FP));
  p = (FP *) calloc(N, sizeof(FP));
  r = (FP *) calloc(N, sizeof(FP));
  q = (FP *) calloc(N, sizeof(FP));
  temp = (FP *) calloc(N, sizeof(FP));

  initialize(A, row, col, b, n, N);

  if (argc==3) {
    gpunum = atoi(argv[2]); // Device number
    if ((gpunum > 2) || (gpunum < 0)) {
      printf("Error, Device number must be 0, 1, or 2\n");
      exit (-1);
    }
  }
  hipSetDevice(gpunum);
  printf("Using device %d\n",gpunum);

  // ------------- COMPUTATION DONE ON GPU ----------------------------
  hipEventCreate(&start); // instrument code to measure start time
  hipEventCreate(&stop);
  
  hipEventRecord(start, 0);


  printf("Setting up cuda memory\n");
  hipMalloc((void**)&dev_A, Nz * sizeof(FP)); // allocate memory on device
  hipMalloc((void**)&dev_row, (N+1) * sizeof(int));
  hipMalloc((void**)&dev_col, Nz * sizeof(int));
  hipMalloc((void**)&dev_b, N * sizeof(FP));
  hipMalloc((void**)&dev_x, N * sizeof(FP));
  hipMalloc((void**)&dev_p, N * sizeof(FP));
  hipMalloc((void**)&dev_r, N * sizeof(FP));
  hipMalloc((void**)&dev_q, N * sizeof(FP));
  hipMalloc((void**)&dev_temp, N * sizeof(FP));



  printf("Copy from host to cuda memory\n");
  hipMemcpy(dev_A, A , Nz * sizeof(FP), hipMemcpyHostToDevice);
  hipMemcpy(dev_row, row , (N+1) * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_col, col , Nz * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b , N * sizeof(FP), hipMemcpyHostToDevice);
  hipMemcpy(dev_x, x , N * sizeof(FP), hipMemcpyHostToDevice);
  hipMemcpy(dev_p, p , N * sizeof(FP), hipMemcpyHostToDevice);
  hipMemcpy(dev_r, r , N * sizeof(FP), hipMemcpyHostToDevice);
  hipMemcpy(dev_q, q , N * sizeof(FP), hipMemcpyHostToDevice);
  hipMemcpy(dev_temp, temp, N * sizeof(FP), hipMemcpyHostToDevice);

  // const FP THRESHOLD = sqrt(vecdot(b, b, N)) / 1000000.;
  gpu_vecdot<<<N/BDIM, BDIM>>>(dev_b, dev_b, dev_temp, N);
  hipMemcpy(temp,dev_temp, N * sizeof(FP), hipMemcpyDeviceToHost);
  const FP THRESHOLD = sqrt(reduce(temp, N)) / 1000000.;

  gpu_matmulvec<<<N/BDIM, BDIM>>>(dev_A, dev_row, dev_col, dev_x, dev_r, Nz);

  gpu_vecop<<<N/BDIM, BDIM>>>(dev_b, dev_r, dev_r, -1., N);
  gpu_vecdot<<<N/BDIM, BDIM>>>(dev_r, dev_r, dev_temp, N);
  hipMemcpy(temp,dev_temp, N * sizeof(FP), hipMemcpyDeviceToHost);
  rho = reduce(temp, N);

  printf("Stopping tolerance: %.6e\n", THRESHOLD);
  printf("Initial residuals: %.6e\n", sqrt(rho));

  for(iter = 0; iter < MAX_ITERATION && sqrt(rho) >= THRESHOLD; iter++) {
    prev_rho = rho;
    gpu_vecdot<<<N/BDIM, BDIM>>>(dev_r, dev_r, dev_temp, N);
    hipMemcpy(temp,dev_temp, N * sizeof(FP), hipMemcpyDeviceToHost);
    rho = reduce(temp, N);
    // printf("current residuals: %f\n", sqrt(rho));
    if(iter == 0) {
      gpu_vecassign<<<N/BDIM, BDIM>>>(dev_p, dev_r, N);
    } else {
        FP beta = rho / prev_rho;
        // printf("beta: %.6e\n", beta);
        gpu_vecop<<<N/BDIM, BDIM>>>(dev_r, dev_p, dev_p, beta, N);
    }
    // q = Ap
    gpu_vecreset<<<N/BDIM, BDIM>>>(dev_q, N);
    gpu_matmulvec<<<N/BDIM, BDIM>>>(dev_A, dev_row, dev_col, dev_p, dev_q, Nz);
    //a = pTq
    gpu_vecdot<<<N/BDIM, BDIM>>>(dev_p, dev_q, dev_temp, N);
    hipMemcpy(temp,dev_temp, N * sizeof(FP), hipMemcpyDeviceToHost);
    FP alpha = reduce(temp, N);
    // printf("alpha: %.6e\n", alpha);
    // x = x + ap
    gpu_vecop<<<N/BDIM, BDIM>>>(dev_x, dev_p, dev_x, alpha, N);
    // vecop(x, p, x, alpha, N);
    // r = r -aq
    gpu_vecop<<<N/BDIM, BDIM>>>(dev_r, dev_q, dev_r, -alpha, N);
    // vecop(r, q, r, -alpha, N);

  }
  hipEventRecord(stop, 0); // instrument code to measure end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop );

  printf("#CG iterations: %d\n", iter);
  printf("Final residuals: %.6e\n", sqrt(rho));
  printf("Time to calculate results on GPU: %f seconds.\n", elapsed_time_ms/1000.); 





// -------------- clean up ---------------------------------------

  free(A);
  free(row);
  free(col);
  free(b);
  free(x);
  free(p);
  free(r);
  free(q);
  hipFree(dev_A);
  hipFree(dev_row);
  hipFree(dev_col);
  hipFree(dev_b);
  hipFree(dev_x);
  hipFree(dev_p);
  hipFree(dev_r);
  hipFree(dev_q);
  hipFree(dev_temp);

  // cudaEventDestroy(start);
  // cudaEventDestroy(stop);

  return 0;
}
