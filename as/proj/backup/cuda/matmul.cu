#include "hip/hip_runtime.h"
#define FP float
#BDIM 1024
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>


__global__ void gpu_vecop(FP *a, FP *b, FP *c, FP coef, int N) {
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  if(index < N) c[index] = a[index] + coef * b[index];
}

__global__ void gpu_vecdot(FP *a, FP *b, FP *c, int N) {
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  if(index < N) c[index] = a[index] * b[index]; 
}


__global__ void gpu_matmulvec(FP *a, int *row, int *col, FP *b, FP *c, int Nz) {

  int col = threadIdx.x + blockDim.x * blockIdx.x;
  int row = threadIdx.y + blockDim.y * blockIdx.y;

  int indexb = col;
  int index = row * m + col;
  
  if(col < m && row < n) {
    c[index] = 0.;
    for (int indexa = row*p; indexa < (row*p + p); indexa++, indexb+=m) 
      c[index] += a[indexa]*b[indexb];
  }

}

void print(FP *a, int n, int m) {
	for(int i=0; i<n; i++) {
		for(int j=0; j<m; j++) {
			printf("%.6e ", a[i*m+j]);
		}
		printf("\n");
	}
}

void kij(FP *a, FP *b, FP *c, int n, int p, int m) {
  for(int k = 0; k < p; k++) {
    for(int i = 0; i < n; i++) {
	  //FP r = a[i][k];
	  FP r = a[i * p + k];
	  const int baseC = i * m;
	  const int baseB = k * m;
      for(int j = 0; j < m; j++) 
        c[baseC + j] -= r * b[baseB + j];
    }
  }
}

void cpu_matrixmult(FP *a,FP *b, FP *c, int n) {

  int index, indexa, indexb;
  FP cvalue;
  for(int col=0;col < n; col++)
    for(int row=0;row < n; row++) {
      indexb = col;
      index = row * n + col;
      cvalue = 0.;
      for (indexa = row*n; indexa < (row*n + n); indexa++, indexb+=n) 
	cvalue += a[indexa]*b[indexb];
      c[index] -= cvalue; //NOTE: This calculates the diff between CPU and GPU computations.
    }
}


int main(int argc, char *argv[]) {

  int i, j; // loop counters

  int gpucount = 0; // Count of available GPUs
  int gpunum = 0; // Device number to use
  int Grid_Dim_X = 1, Grid_Dim_Y = 1; //Grid dimension, x and y
  int Block_Dim_X = 1, Block_Dim_Y = 1; //Block dimension, x and y, square

  int n, N; // matrix dimension
  int Nz_in_grid, Nz; 
  float rho, prev_rho;
  int *row, *col;
  FP *A, *b, *x, *p, *r, *q;
  FP *dev_A, *dev_row, *dev_col, *dev_b, *dev_x, *dev_p, *dev_r, *dev_q;

  hipEvent_t start, stop; // using cuda events to measure time
  float elapsed_time_ms; // which is applicable for asynchronous code also
  hipError_t errorcode;

  // --------------------SET PARAMETERS AND DATA -----------------------

  errorcode = hipGetDeviceCount(&gpucount);
  if (errorcode == hipErrorNoDevice) {
    printf("No GPUs are visible\n");
    exit(-1);
  }
  else {
     printf("Device count = %d\n",gpucount);
  }

  if ((argc<2) || (argc>3)) {
    printf("Usage: matmul <matrix dim n> [<dev num>]\n");
    exit (-1);
  }

  n = atoi(argv[1]);
  N = n * n;
  Nz_in_grid = (n - 2) * 3 + 2 * 2; // number of non-zero elements in a grid
  Nz = (n - 2) * 3 * Nz_in_grid + 2 * 2 * Nz_in_grid; // number of non-zero elements
  row = (int *) calloc(Nz, sizeof(int));
  col = (int *) calloc(Nz, sizeof(int));
  A = (FP *) calloc(Nz, sizeof(FP));
  b = (FP *) calloc(N, sizeof(FP));
  x = (FP *) calloc(N, sizeof(FP));
  p = (FP *) calloc(N, sizeof(FP));
  r = (FP *) calloc(N, sizeof(FP));
  q = (FP *) calloc(N, sizeof(FP));


  // Block_Dim_X = atoi(argv[4]);
  // Block_Dim_Y = atoi(argv[5]);
  // if (Block_Dim_X*Block_Dim_Y > 1024) {
  //   printf("Error, too many threads in block\n");
  //   exit (-1);
  // }

  // Grid_Dim_X = (int) ceil((double) m / Block_Dim_X);
  // Grid_Dim_Y = (int) ceil((double) n / Block_Dim_Y);
  // if (Grid_Dim_X * Grid_Dim_Y * Block_Dim_X * Block_Dim_Y < n * m) {
  //   printf("Error, number of threads in x/y dimensions less than number of array elements\n");
  //   exit (-1);
  // }

  if (argc==3) {
    gpunum = atoi(argv[2]); // Device number
    if ((gpunum > 2) || (gpunum < 0)) {
      printf("Error, Device number must be 0, 1, or 2\n");
      exit (-1);
    }
  }
  hipSetDevice(gpunum);
  printf("Using device %d\n",gpunum);
  
  // printf("Matrix Dimension = %d\n",n);
  // printf("Block_Dim_X = %d, Block_Dim_Y = %d, Grid_Dim_X = %d, Grid_Dim_Y = %d\n",Block_Dim_X, Block_Dim_Y, Grid_Dim_X, Grid_Dim_Y);

  // dim3 Grid(Grid_Dim_X, Grid_Dim_Y); //Grid structure
  // dim3 Block(Block_Dim_X, Block_Dim_Y); //Block structure



  // ------------- COMPUTATION DONE ON GPU ----------------------------


  hipMalloc((void**)&dev_A, Nz * sizeof(FP)); // allocate memory on device
  hipMalloc((void**)&dev_row, Nz * sizeof(int));
  hipMalloc((void**)&dev_col, Nz * sizeof(int));
  hipMalloc((void**)&dev_b, p * m * sizeof(FP));
  hipMalloc((void**)&dev_x, n * m * sizeof(FP));
  hipMalloc((void**)&dev_p, n * m * sizeof(FP));
  hipMalloc((void**)&dev_r, n * m * sizeof(FP));
  hipMalloc((void**)&dev_q, n * m * sizeof(FP));



  hipMemcpy(dev_A, A , Nz * sizeof(FP), hipMemcpyHostToDevice);
  hipMemcpy(dev_row, row , Nz * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_col, col , Nz * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b , N * sizeof(FP), hipMemcpyHostToDevice);
  hipMemcpy(dev_x, x , N * sizeof(FP), hipMemcpyHostToDevice);
  hipMemcpy(dev_p, p , N * sizeof(FP), hipMemcpyHostToDevice);
  hipMemcpy(dev_r, r , N * sizeof(FP), hipMemcpyHostToDevice);
  hipMemcpy(dev_q, q , N * sizeof(FP), hipMemcpyHostToDevice);
  

  // gpu_matrixmult<<<Grid,Block>>>(dev_a, dev_b, dev_c, n, p, m);
  gpu_matmulvec<<<N/BDIM, BDIM>>>(dev_A, dev_row, dev_col, dev_x, dev_r, Nz);
  gpu_vecop<<<N/BDIM, BDIM>>>(dev_b, dev_r, dev_r, -1., N);
  hipMemcpy(r,dev_r, N * sizeof(FP), hipMemcpyDeviceToHost);


  // // r = b - A * x0
  // matmulvec(Nz, A, row, col, x, r, N);
  // vecop(b, r, r, -1., N);
  // rho = vecdot(r, r, N);

  // for (iter = 0; iter < MAX_ITERATION && sqrt(rho) >= THRESHOLD; iter++) {
  //     // r =rTr
  //     prev_rho = rho;
  //     rho = vecdot(r, r, N);
  //     //printf("current residuals: %f\n", sqrt(rho));
  //     if (iter == 0) {
  //         // p = r;
  //         for (int i = 0; i < N; i++) {
  //             p[i] = r[i];
  //         }
  //     } else {
  //         // beta = rho_p / rho_p-1
  //         FP beta = rho / prev_rho;
  //         //printf("beta: %f\n", beta);
  //         // p = r + beta * p;
  //         vecop(r, p, p, beta, N);
  //     }
  //     // q = Ap
  //     for (int j = 0; j < N; j++) q[j] = .0;
  //     matmulvec(Nz, A, row, col, p, q, N);
  //     //a = pTq
  //     FP alpha = vecdot(p, q, N);
  //     //printf("alpha: %f\n", alpha);
  //     // x = x + ap
  //     vecop(x, p, x, alpha, N);
  //     // r = r -aq
  //     vecop(r, q, r, -alpha, N);

  // }


  // hipMemcpy(dev_a, a , n * p * sizeof(FP), hipMemcpyHostToDevice);
  // hipMemcpy(dev_b, b , p * m * sizeof(FP), hipMemcpyHostToDevice);

  // gpu_matrixmult<<<Grid,Block>>>(dev_a, dev_b, dev_c, n, p, m);

  // hipMemcpy(c,dev_c, n * m * sizeof(FP), hipMemcpyDeviceToHost);



// -------------- clean up ---------------------------------------

  free(A);
  free(row);
  free(col);
  free(b);
  free(x);
  free(p);
  free(r);
  free(q);
  hipFree(dev_A);
  hipFree(dev_row);
  hipFree(dev_col);
  hipFree(dev_b);
  hipFree(dev_x);
  hipFree(dev_p);
  hipFree(dev_r);
  hipFree(dev_q);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
