#define FP double

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>

__global__ void gpu_matrixmult(FP *a,FP *b, FP *c, int n, int p, int m) {

  int col = threadIdx.x + blockDim.x * blockIdx.x;
  int row = threadIdx.y + blockDim.y * blockIdx.y;

  int indexb = col;
  int index = row * m + col;
  
  if(col < m && row < n) {
    c[index] = 0.;
    for (int indexa = row*p; indexa < (row*p + p); indexa++, indexb+=m) 
      c[index] += a[indexa]*b[indexb];
  }

}

void print(FP *a, int n, int m) {
	for(int i=0; i<n; i++) {
		for(int j=0; j<m; j++) {
			printf("%.6e ", a[i*m+j]);
		}
		printf("\n");
	}
}

void kij(FP *a, FP *b, FP *c, int n, int p, int m) {
  for(int k = 0; k < p; k++) {
    for(int i = 0; i < n; i++) {
	  //FP r = a[i][k];
	  FP r = a[i * p + k];
	  const int baseC = i * m;
	  const int baseB = k * m;
      for(int j = 0; j < m; j++) 
        c[baseC + j] -= r * b[baseB + j];
    }
  }
}

void cpu_matrixmult(FP *a,FP *b, FP *c, int n) {

  int index, indexa, indexb;
  FP cvalue;
  for(int col=0;col < n; col++)
    for(int row=0;row < n; row++) {
      indexb = col;
      index = row * n + col;
      cvalue = 0.;
      for (indexa = row*n; indexa < (row*n + n); indexa++, indexb+=n) 
	cvalue += a[indexa]*b[indexb];
      c[index] -= cvalue; //NOTE: This calculates the diff between CPU and GPU computations.
    }
}


int main(int argc, char *argv[]) {

  int i, j; // loop counters

  int gpucount = 0; // Count of available GPUs
  int gpunum = 0; // Device number to use
  int Grid_Dim_X = 1, Grid_Dim_Y = 1; //Grid dimension, x and y
  int Block_Dim = 1; //Block dimension, x and y, square

  int n, p, m; // matrix dimension
  FP *a,*b,*c;
  FP *dev_a, *dev_b, *dev_c;

  hipEvent_t start, stop; // using cuda events to measure time
  float elapsed_time_ms; // which is applicable for asynchronous code also
  hipError_t errorcode;

  // --------------------SET PARAMETERS AND DATA -----------------------

  errorcode = hipGetDeviceCount(&gpucount);
  if (errorcode == hipErrorNoDevice) {
    printf("No GPUs are visible\n");
    exit(-1);
  }
  else {
     printf("Device count = %d\n",gpucount);
  }

  if ((argc<7) || (argc>8)) {
    printf("Usage: matmul <matrix dim n> <matrix dim p> <matrix dim m> <block dim> <grid dim x> <grid dim y> [<dev num>]\n");
    exit (-1);
  }

  n = atoi(argv[1]);
  p = atoi(argv[2]);
  m = atoi(argv[3]);

  Block_Dim = atoi(argv[4]); // Square block
  if (Block_Dim*Block_Dim > 1024) {
    printf("Error, too many threads in block\n");
    exit (-1);
  }

  Grid_Dim_X = atoi(argv[5]);
  Grid_Dim_Y = atoi(argv[6]);
  if (Grid_Dim_X * Grid_Dim_Y * Block_Dim * Block_Dim < n * m) {
    printf("Error, number of threads in x/y dimensions less than number of array elements\n");
    exit (-1);
  }

  if (argc==8) {
    gpunum = atoi(argv[7]); // Device number
    if ((gpunum > 2) || (gpunum < 0)) {
      printf("Error, Device number must be 0, 1, or 2\n");
      exit (-1);
    }
  }
  hipSetDevice(gpunum);
  printf("Using device %d\n",gpunum);
  
  printf("Matrix Dimension = %d\n",n);
  printf("Block_Dim = %d, Grid_Dim_X = %d, Grid_Dim_Y = %d\n",Block_Dim,Grid_Dim_X, Grid_Dim_Y);

  dim3 Grid(Grid_Dim_X, Grid_Dim_Y); //Grid structure
  dim3 Block(Block_Dim, Block_Dim); //Block structure


  a = (FP*) malloc(n * p * sizeof(FP)); // dynamically allocated memory for arrays on host
  b = (FP*) malloc(p * m * sizeof(FP));
  c = (FP*) malloc(n * m * sizeof(FP)); // results from GPU
  srand(12345);
  for(i=0; i<n; i++)
    for(j=0; j < p; j++) {
      a[i * p + j] = (FP) rand() / (FP) RAND_MAX;
      //      a[i * p + j] = (FP) i+j; // may be helpful for debugging
    }

  for(i=0; i<p; i++)
    for(j=0; j<m; j++) {
      b[i * m + j] = (FP) rand() / (FP) RAND_MAX;
      //      b[i * n + j] = (FP) i+j; // may be helpful for debugging
    }

  // ------------- COMPUTATION DONE ON GPU ----------------------------

  hipMalloc((void**)&dev_a, n * p * sizeof(FP)); // allocate memory on device
  hipMalloc((void**)&dev_b, p * m * sizeof(FP));
  hipMalloc((void**)&dev_c, n * m * sizeof(FP));

  hipMemcpy(dev_a, a , n * p * sizeof(FP), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b , p * m * sizeof(FP), hipMemcpyHostToDevice);

  hipEventCreate(&start); // instrument code to measure start time
  hipEventCreate(&stop);
  
  hipEventRecord(start, 0);
  // cudaEventSynchronize(start); // not needed

  gpu_matrixmult<<<Grid,Block>>>(dev_a, dev_b, dev_c, n, p, m);

  hipEventRecord(stop, 0); // instrument code to measure end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop );

  hipMemcpy(c,dev_c, n * m * sizeof(FP), hipMemcpyDeviceToHost);

  printf("Time to calculate results on GPU: %f ms.\n", elapsed_time_ms); // exec. time

  // ------------- COMPUTATION DONE ON HOST CPU ----------------------------
  // DEBUGGING USE ONLY (AND FOR LIMITED NUMBERS OF TIMING RUNS)

  hipEventRecord(start, 0); // use same timing
  // cudaEventSynchronize(start); // not needed


  //cpu_matrixmult(a,b,c, n); // do calculation on host (NOTE: This computes the diff with GPU result.)
  //kij(a, b, c, n, p, m); // 

  hipEventRecord(stop, 0); // instrument code to measue end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop );

  printf("Time to calculate results on CPU: %f ms.\n", elapsed_time_ms); // exec. time
  //print(c, n, m);
// ------------------- check device creates correct results -----------------

  double error, suma, sumb, sumc, ai, bi, ci;
  suma = 0.; sumb = 0; sumc = 0;
  for(i=0; i<n*p; i++) {
    ai = (double) a[i];
    suma += ai * ai;
  }
  for(i=0; i<p*m; i++) {
	bi = (double) b[i];
    sumb += bi * bi;
  }
  for(i=0; i<n*m; i++) {
	ci = (double) c[i];
    sumc += ci * ci;
  }
  suma = sqrt(suma);
  sumb = sqrt(sumb);
  sumc = sqrt(sumc);
  error =  sumc/(n*suma*sumb);
  printf("Scaled error between GPU and CPU: %e\n", error);

// -------------- clean up ---------------------------------------

  free(a);
  free(b);
  free(c);
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
