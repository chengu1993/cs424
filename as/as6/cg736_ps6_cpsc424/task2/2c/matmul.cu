#define FP float
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>

/* Utility function
 * print out matrix 
 * for debug use only*/
void print(FP *a, int n, int m) {
	for(int i=0; i<n; i++) {
		for(int j=0; j<m; j++) {
			printf("%.6e ", a[i*m+j]);
		}
		printf("\n");
	}
}
__global__ void gpu_matrixmult(FP *a,FP *b, FP *c, int n, int p, int m, int TW) {

  extern __shared__ FP bigarray[];
  FP *atile = &bigarray[0], *btile = &bigarray[TW*TW];

  //__shared__ FP atile[TW][TW], btile[TW][TW];

  int tx = threadIdx.x, ty = threadIdx.y;
  int col = tx + blockDim.x * blockIdx.x;
  int row = ty + blockDim.y * blockIdx.y;
  FP cvalue = 0.;
  int tile_num = (int) ceil((double)p/TW);
  //printf("ceil:%d\n", tile_num);
   
	// full size tile
	for(int i=0; i<tile_num; i++) {
      //atile[ty][tx] = a[row*p + i*TW + tx]; //load elements into atile
	  //btile[ty][tx] = b[(i*TW+ty)*m + col];  //load elements into btile
	  atile[ty*TW+tx] = a[row*p + i*TW + tx];
      btile[ty*TW+tx] = b[(i*TW+ty)*m + col];
	  __syncthreads();

	  int boarder = ((p%TW) !=0 && i == tile_num-1) ? p % TW : TW; 
	  for(int indexa=ty*TW, indexb=tx; indexa<ty*TW+boarder; indexa++,indexb+=TW) {
	    cvalue += atile[indexa] * btile[indexb];
	  }
	  //for(int j=0; j<boarder; j++) {
	    //cvalue += atile[ty][j] * btile[j][tx];
	  //}
	  __syncthreads();
     
    }
    if(row < n && col < m){
      c[row * m + col] = cvalue;
    }
}


void kij(FP *a, FP *b, FP *c, int n, int p, int m) {
  for(int k = 0; k < p; k++) {
    for(int i = 0; i < n; i++) {
	  //FP r = a[i][k];
	  FP r = a[i * p + k];
	  const int baseC = i * m;
	  const int baseB = k * m;
      for(int j = 0; j < m; j++) 
        c[baseC + j] -= r * b[baseB + j];
    }
  }
}

void cpu_matrixmult(FP *a,FP *b, FP *c, int n) {

  int index, indexa, indexb;
  FP cvalue;
  for(int col=0;col < n; col++)
    for(int row=0;row < n; row++) {
      indexb = col;
      index = row * n + col;
      cvalue = 0.;
      for (indexa = row*n; indexa < (row*n + n); indexa++, indexb+=n) 
	cvalue += a[indexa]*b[indexb];
      c[index] -= cvalue; //NOTE: This calculates the diff between CPU and GPU computations.
    }
}


int main(int argc, char *argv[]) {

  int i, j; // loop counters

  int gpucount = 0; // Count of available GPUs
  int gpunum = 0; // Device number to use
  int Grid_Dim_X = 1, Grid_Dim_Y = 1; //Grid dimension, x and y
  int Block_Dim_X = 1, Block_Dim_Y = 1; //Block dimension, x and y, square

  int n, p, m; // matrix dimension
  FP *a,*b,*c;
  FP *dev_a, *dev_b, *dev_c;

  hipEvent_t start, stop; // using cuda events to measure time
  float elapsed_time_ms; // which is applicable for asynchronous code also
  hipError_t errorcode;

  // --------------------SET PARAMETERS AND DATA -----------------------

  errorcode = hipGetDeviceCount(&gpucount);
  if (errorcode == hipErrorNoDevice) {
    printf("No GPUs are visible\n");
    exit(-1);
  }
  else {
     printf("Device count = %d\n",gpucount);
  }

  if ((argc<6) || (argc>7)) {
    printf("Usage: matmul <matrix dim n> <matrix dim p> <matrix dim m> <block dim x> <block dim y> [<dev num>]\n");
    exit (-1);
  }

  n = atoi(argv[1]);
  p = atoi(argv[2]);
  m = atoi(argv[3]);

  Block_Dim_X = atoi(argv[4]);
  Block_Dim_Y = atoi(argv[5]);
  if (Block_Dim_X*Block_Dim_Y > 1024) {
    printf("Error, too many threads in block\n");
    exit (-1);
  }

  Grid_Dim_X = (int) ceil((double) m / Block_Dim_X);
  Grid_Dim_Y = (int) ceil((double) n / Block_Dim_Y);
  //printf("Dimx %d\n",Grid_Dim_X);
  //printf("Dimy %d\n",Grid_Dim_Y);
  if (Grid_Dim_X * Grid_Dim_Y * Block_Dim_X * Block_Dim_Y < n * m) {
    printf("Error, number of threads in x/y dimensions less than number of array elements\n");
    exit (-1);
  }

  if (argc==7) {
    gpunum = atoi(argv[6]); // Device number
    if ((gpunum > 2) || (gpunum < 0)) {
      printf("Error, Device number must be 0, 1, or 2\n");
      exit (-1);
    }
  }
  hipSetDevice(gpunum);
  printf("Using device %d\n",gpunum);
  
  printf("Matrix Dimension = %d %d %d\n",n, p, m);
  printf("Block_Dim_X = %d, Block_Dim_Y = %d, Grid_Dim_X = %d, Grid_Dim_Y = %d\n", Block_Dim_X, Block_Dim_Y, Grid_Dim_X, Grid_Dim_Y);

  dim3 Grid(Grid_Dim_X, Grid_Dim_Y); //Grid structure
  dim3 Block(Block_Dim_X, Block_Dim_Y); //Block structure


  a = (FP*) malloc(n * p * sizeof(FP)); // dynamically allocated memory for arrays on host
  b = (FP*) malloc(p * m * sizeof(FP));
  c = (FP*) malloc(n * m * sizeof(FP)); // results from GPU
  srand(12345);
  for(i=0; i<n; i++)
    for(j=0; j < p; j++) {
      a[i * p + j] = (FP) rand() / (FP) RAND_MAX;
      //      a[i * p + j] = (FP) i+j; // may be helpful for debugging
    }

  for(i=0; i<p; i++)
    for(j=0; j<m; j++) {
      b[i * m + j] = (FP) rand() / (FP) RAND_MAX;
      //      b[i * n + j] = (FP) i+j; // may be helpful for debugging
    }

  //printf("A\n");
  //print(a, n, p);
  //printf("B\n");
  //print(b, p, m);

  // ------------- COMPUTATION DONE ON GPU ----------------------------

  hipMalloc((void**)&dev_a, n * p * sizeof(FP)); // allocate memory on device
  hipMalloc((void**)&dev_b, p * m * sizeof(FP));
  hipMalloc((void**)&dev_c, n * m * sizeof(FP));

  hipMemcpy(dev_a, a , n * p * sizeof(FP), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b , p * m * sizeof(FP), hipMemcpyHostToDevice);

  hipEventCreate(&start); // instrument code to measure start time
  hipEventCreate(&stop);
  
  hipEventRecord(start, 0);
  // cudaEventSynchronize(start); // not needed
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(gpu_matrixmult), hipFuncCachePreferShared);
  const int TW = Block_Dim_X; 
  size_t Ns = 2 * TW * TW * sizeof(FP);
  gpu_matrixmult<<<Grid,Block, Ns>>>(dev_a, dev_b, dev_c, n, p, m, TW);
  hipEventRecord(stop, 0); // instrument code to measure end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop );

  hipMemcpy(c,dev_c, n * m * sizeof(FP), hipMemcpyDeviceToHost);

  printf("Time to calculate results on GPU: %f ms.\n", elapsed_time_ms); // exec. time

  //print(c, n, m);
  // ------------- COMPUTATION DONE ON HOST CPU ----------------------------
  // DEBUGGING USE ONLY (AND FOR LIMITED NUMBERS OF TIMING RUNS)

  hipEventRecord(start, 0); // use same timing
  // cudaEventSynchronize(start); // not needed


  //cpu_matrixmult(a,b,c, n); // do calculation on host (NOTE: This computes the diff with GPU result.)
  //kij(a, b, c, n, p, m); // 

  hipEventRecord(stop, 0); // instrument code to measue end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop );

  printf("Time to calculate results on CPU: %f ms.\n", elapsed_time_ms); // exec. time
  //print(c, n, m);
// ------------------- check device creates correct results -----------------

  double error, suma, sumb, sumc, ai, bi, ci;
  suma = 0.; sumb = 0; sumc = 0;
  for(i=0; i<n*p; i++) {
    ai = (double) a[i];
    suma += ai * ai;
  }
  for(i=0; i<p*m; i++) {
	bi = (double) b[i];
    sumb += bi * bi;
  }
  for(i=0; i<n*m; i++) {
	ci = (double) c[i];
    sumc += ci * ci;
  }
  suma = sqrt(suma);
  sumb = sqrt(sumb);
  sumc = sqrt(sumc);
  error =  sumc/(sqrt(n*m)*suma*sumb);
  printf("Scaled error between GPU and CPU: %e\n", error);

// -------------- clean up ---------------------------------------

  free(a);
  free(b);
  free(c);
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
